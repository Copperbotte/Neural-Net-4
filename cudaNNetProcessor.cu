#include "hip/hip_runtime.h"

#include "OGLWindow.h"
#include "NNet.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include "cudaNNetProcessor.cuh"

int clamp(int x, int low, int high)
{
    if (x < low) x = low;
    if (high < x) x = high;
    return x;
}

//__constant__ int cuShapeLen[1];
//__constant__ int* cuSigmoidIndices; //hardcoded: 0 is const, 1 is tanh

hipError_t cudaNNetProcessor::makeBuffers()
{
    // error check dependent pointers
    if (!_pNet)
    {
        std::cout << "Neural Net not initialized!\n";
        return hipErrorNotInitialized;
    }

    //initialize cuda
    hipError_t cudaStatus = hipSuccess;

    //shape length
    _numWeights = _pNet->getShapeLen() - 1;
    //cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(_cuNumWeights), &_numWeights, sizeof(int), 0, hipMemcpyHostToDevice);
    //can __constant__ be put in a class?
    cudaStatus = hipMalloc((void**)&_cuNumWeights, sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        makeMallocError("weight length", cudaStatus);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(_cuNumWeights, &_numWeights, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        makeMemcpyError("weight length", cudaStatus);
        return cudaStatus;
    }

    /*
    //copy sigmoid lookup (hardcoded)
    //The neural net on cuda is already pretty slow.
    //This will add additional overhead that isn't nessisary right now.
    int* cuSigmoidIndices;
    cudaStatus = hipMalloc((void**)&cuSigmoidIndices, netshapelen * sizeof(int));
    int *SigmoidIndices = new int[netshapelen];
    for (int i = 1; i < netshapelen; ++i)
        SigmoidIndices[i] = 1;
    SigmoidIndices[0] = 0;
    cudaStatus = hipMemcpy(cuSigmoidIndices, SigmoidIndices, netshapelen * sizeof(int), hipMemcpyHostToDevice);
    */

    //initialize weights
    _cuWeightData = new float* [_numWeights];
    cudaStatus = hipMalloc((void**)&_cuWeights, _numWeights * sizeof(cuMatrix));
    if (cudaStatus != hipSuccess)
    {
        makeMallocError("neural network weights", cudaStatus);
        return cudaStatus;
    }

    for (int i = 0; i < _numWeights; ++i)
    {
        cuMatrix temp = cuMatrix(_pNet->getWeights(i));
        cudaStatus = hipMemcpy(&_cuWeights[i], &temp, sizeof(cuMatrix), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            std::string name = "weight matrix " + std::to_string(i);
            makeMemcpyError(name.c_str(), cudaStatus);
            return cudaStatus;
        }

        float* tempData;
        size_t tempDataSize = temp.cols * temp.rows * sizeof(float);
        cudaStatus = hipMalloc((void**)&tempData, tempDataSize);
        if (cudaStatus != hipSuccess)
        {
            std::string name = "weight matrix " + std::to_string(i) + " data";
            makeMallocError(name.c_str(), cudaStatus);
            return cudaStatus;
        }

        cudaStatus = hipMemcpy(tempData, temp.data, tempDataSize, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            std::string name = "weight matrix " + std::to_string(i) + " data";
            makeMemcpyError(name.c_str(), cudaStatus);
            return cudaStatus;
        }

        cudaStatus = hipMemcpy(&(_cuWeights[i].data), &tempData, sizeof(float*), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            std::string name = "weight matrix " + std::to_string(i) + " data pointer";
            makeMemcpyError(name.c_str(), cudaStatus);
            return cudaStatus;
        }

        _cuWeightData[i] = tempData;
    }

    return cudaStatus;
}

void cudaNNetProcessor::makeMallocError(const char* err, hipError_t cudaStatus) const
{
    std::cout << "Failed to allocate " << err << " on gpu!\n";
    std::cout << "error code: " << hipGetErrorString(cudaStatus);
}

void cudaNNetProcessor::makeMemcpyError(const char* err, hipError_t cudaStatus) const
{
    std::cout << "Failed to copy " << err << " to gpu!\n";
    std::cout << "error code: " << hipGetErrorString(cudaStatus);
}

void cudaNNetProcessor::safeFree(void** ptr)
{
    if (!*ptr) return;
    hipFree(*ptr);
    *ptr = nullptr;
}

cudaNNetProcessor::cudaNNetProcessor() :
    _cuWeights(nullptr), _cuWeightData(nullptr),
    _numWeights(0), _pNet(nullptr)
{
}

cudaNNetProcessor::~cudaNNetProcessor()
{
    for (int i = 0; i < _numWeights; ++i)
        safeFree((void**)&_cuWeightData[i]);
    safeFree((void**)&_cuWeightData);

    // These pointers are for reference only, and may not lead anywhere
    _pNet = nullptr;

    // This was copied from an nvidia cuda sample.
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }
}

cudaNNetProcessor::cudaNNetProcessor(NNet& pNet) :
    _cuWeights(nullptr), _cuWeightData(nullptr),
    _numWeights(0), _pNet(&pNet)
{
    makeBuffers();
}

cudaNNetProcessor::cudaNNetProcessor(const cudaNNetProcessor& N) :
    _cuWeights(nullptr), _cuWeightData(nullptr),
    _numWeights(0), _pNet(N._pNet)
{
    makeBuffers();
}

hipError_t cudaNNetProcessor::cudaCopyNNet() const
{
    hipError_t cudaStatus = hipSuccess;
    for (int i = 0; i < _numWeights; ++i)
    {
        cudaStatus = hipMemcpyAsync(_cuWeightData[i], _pNet->getWeights(i).getDataPtr(),
            _pNet->getWeights(i).getCols() * _pNet->getWeights(i).getRows() * sizeof(float),
            hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            std::string name = "weight matrix " + std::to_string(i) + " data";
            makeMemcpyError(name.c_str(), cudaStatus);
            return cudaStatus;
        }
    }
    return cudaStatus;
}
