#include "hip/hip_runtime.h"

#include "OGLWindow.h"
#include "NNet.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

int clamp(int x, int low, int high)
{
    if (x < low) x = low;
    if (high < x) x = high;
    return x;
}

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

struct pass_data
{
    NNet* Net;
    matrix* in;
    matrix* out;
};

int main()
{
    NNet net = NNet();
    net.randomizeNodes();

    float **i_function = new float*[100];
    float* o_function = new float[100];
    matrix* i_matrix = new matrix[100];
    matrix* o_matrix = new matrix[100];
    for (int i = 0; i < 100; ++i)
    {
        i_function[i] = new float[2];
        int x = i % 10;
        int y = i / 10;
        i_function[i][0] = (float)x * (10.0 / 6.0);
        i_function[i][1] = (float)y * (10.0 / 6.0);
        o_function[i] = sin(i_function[i][0]) * sin(i_function[i][1]);
        i_matrix[i] = matrix(1, 2, i_function[i]);
        o_matrix[i] = matrix(1, 1, nullptr);
        o_matrix[i].setData(0, 0, o_function[i]);
    }
    
    /*
    for (int i = 0; i < 1000; ++i)
    {
        //float err = net.backProp(input, expected);
        float err = net.backPropArray(i_matrix, o_matrix, 100);
        std::cout << i << " : " << err << '\n';
    }

    for (int i = 0; i < 100; ++i)
        delete[] i_function[i];
    delete[] o_function;
    delete[] i_matrix;
    delete[] o_matrix;
    */
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    OGLWindow wnd("NN4");

    pass_data p;
    p.Net = &net;
    p.in = i_matrix;
    p.out = o_matrix;

    wnd.extra = &p;

    wnd.setPrintFunc([](const char* str) {std::cout << str << '\n'; });
    wnd.fillColorBuffer(0xFF, 0x8F, 0x00, 0xFF);

    GLFWwindow* window = wnd.init();
    wnd.setThinkFunc([](OGLWindow* This, double time)
        {
            GLFWwindow* window = This->getWindowPtr();
            
            pass_data *p = (pass_data*)This->extra;
            NNet* Net = p->Net;

            float err = Net->backPropArray(p->in, p->out, 100);
            std::cout << err << '\n';

            int H = This->getHeight();
            int W = This->getWidth();
            unsigned char *P = This->getColorBufferPtr();

            for (int y = 0; y < H; ++y)
            {
                for (int x = 0; x < W; ++x)
                {
                    unsigned char *c = (y*W + x)*4 + P;
                    float X = 2.0 * (float)x / (float)W;
                    float Y = 2.0 * (float)y / (float)H;
                    float lpi_sample[] = { X,Y };
                    matrix lpinput = matrix(1, 2, lpi_sample);
                    float out = Net->forwardProp(lpinput).getData(0,0);
                    out = tanh(out);
                    out = (out + 1.0) / 2.0;
                    unsigned char color = (int)(out * 255.0f);
                    c[0] = color;
                    c[1] = color;
                    c[2] = color;
                    c[3] = 0xFF;
                }
            }

            //double mx, my;
            //glfwGetCursorPos(window, &mx, &my);

            //my = height - my;

            //mx = clamp(mx, 0, width - 1);
            //my = clamp(my, 0, height - 1);

            //float fm[2] = { (float)mx / width, (float)my / height };
        });
    wnd.think();

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
